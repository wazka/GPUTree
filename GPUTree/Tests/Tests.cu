#include "Tests.cuh"

#include "hip/hip_runtime.h"

int ALL_TESTS = 0;
int PASSED_TESTS = 0;

hipEvent_t start;
hipEvent_t stop;

hipEvent_t startPack;
hipEvent_t stopPack;


void InitAllTests()
{
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	ALL_TESTS = 0;
    PASSED_TESTS = 0;
}

void InitPack()
{
	hipEventCreate(&startPack);
	hipEventCreate(&stopPack);
	hipEventRecord(startPack, 0);
}

void TestInit(const char *testName)
{
	printf("%135s  ", testName);
	++ALL_TESTS;
}

void Assert(bool arg1, bool arg2, bool arg3, bool arg4, bool arg5, bool arg6, bool arg7, bool arg8, bool arg9, bool arg10)
{

	if (arg1 && arg2 && arg3 && arg4 && arg5
		&& arg6 && arg7 && arg8 && arg9 && arg10)
	{
		++PASSED_TESTS;
		printf("   PASS\n");
	}
	else
	{
		printf("   FAILED\n");
	}
}

void Output(std::string output)
{
	printf("\nTest output:\n%s\n", output.c_str());
}

void FinalReport()
{
	float time;
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	printf("===== FINAL REPORT =====\n\n");
	printf("\nPASSED %d / %d\n", PASSED_TESTS, ALL_TESTS);
	printf("ELAPSED TIME: %f\n\n", time);
}

void PackReport()
{
	float time;
	hipEventRecord(stopPack, 0);
	hipEventSynchronize(stopPack);
	hipEventElapsedTime(&time, startPack, stopPack);

	printf("ELAPSED TIME: %f\n\n", time);
}

void PrintException(std::exception e)
{
	printf("   EXCEPTION: %s\n", e.what());
}

void PrintException()
{
	printf("   EXCEPTION\n");
}
